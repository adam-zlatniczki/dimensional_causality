#include "hip/hip_runtime.h"
#include <iostream>
#include "embedding.cuh"

using namespace std;

bool floats_equal(double a, double b, double epsilon = 0.000001)
{
	return fabs(a - b) < epsilon;
}

int test_single_embedding() {
	bool match = true;

	float host_x[10] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0 };
	float host_y[10] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0 };

	float expected_state_space[6 * 3] = {
		5.0, 3.0, 1.0,
		6.0, 4.0, 2.0,
		7.0, 5.0, 3.0,
		8.0, 6.0, 4.0,
		9.0, 7.0, 5.0,
		10.0, 8.0, 6.0 };

	float* dev_state_space_X = 0;
	float* dev_state_space_Y = 0;
	float* dev_state_space_J = 0;
	float* dev_state_space_Z = 0;

	int dev_state_space_size = 0;
	int dev_x_size = 0;

	hipError_t cudaStatus;

	cudaStatus = embed_manifolds(&dev_state_space_X, &dev_state_space_Y, &dev_state_space_J, &dev_state_space_Z, dev_state_space_size, host_x, host_y, 10, dev_x_size, 3, 2, 5);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "embed_manifolds failed!");
		return 1;
	}

	// Copy output vector from GPU buffer to host memory.
	float* host_state_space_X = new float[6 * 3];

	cudaStatus = hipMemcpy(host_state_space_X, dev_state_space_X, 18 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		match = false;
		goto Error;
	}

	for (int i = 0; i < 6*3; i++) {
		if (!floats_equal(expected_state_space[i], host_state_space_X[i])) {
			match = false;
			break;
		}
	}

Error:
	hipFree(dev_state_space_X);
	hipFree(dev_state_space_Y);
	hipFree(dev_state_space_J);
	hipFree(dev_state_space_Z);
	delete[] host_state_space_X;

	if (match) {
		return 0;
	}
	else {
		return 1;
	}
}

int test_joint_embedding() {
	bool match = true;

	float host_x[10] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0 };
	float host_y[10] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0 };

	float expected_state_space[6 * 3] = {
		10.0, 6.0, 2.0,
		12.0, 8.0, 4.0,
		14.0, 10.0, 6.0,
		16.0, 12.0, 8.0,
		18.0, 14.0, 10.0,
		20.0, 16.0, 12.0 };

	float* dev_state_space_X = 0;
	float* dev_state_space_Y = 0;
	float* dev_state_space_J = 0;
	float* dev_state_space_Z = 0;

	int dev_state_space_size = 0;
	int dev_x_size = 0;

	hipError_t cudaStatus;

	cudaStatus = embed_manifolds(&dev_state_space_X, &dev_state_space_Y, &dev_state_space_J, &dev_state_space_Z, dev_state_space_size, host_x, host_y, 10, dev_x_size, 3, 2, 5);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "embed_manifolds failed!");
		return 1;
	}

	// Copy output vector from GPU buffer to host memory.
	float* host_state_space_J = new float[6 * 3];

	cudaStatus = hipMemcpy(host_state_space_J, dev_state_space_J, 18 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		match = false;
		goto Error;
	}

	for (int i = 0; i < 6 * 3; i++) {
		if (!floats_equal(expected_state_space[i], host_state_space_J[i])) {
			match = false;
			break;
		}
	}

Error:
	hipFree(dev_state_space_X);
	hipFree(dev_state_space_Y);
	hipFree(dev_state_space_J);
	hipFree(dev_state_space_Z);
	delete[] host_state_space_J;

	if (match) {
		return 0;
	}
	else {
		return 1;
	}
}

int main() {
	cout << "Running unit tests..." << endl;

	cout << "Test single embedding failed: " << test_single_embedding() << endl;
	cout << "Test joint embedding failed: " << test_joint_embedding() << endl;

	// max data size given the row-like grid-block layout: (2^31-1) * 2^10

	int a = 0;
	cin >> a;
}
