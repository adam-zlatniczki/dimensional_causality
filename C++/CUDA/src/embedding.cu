#include "hip/hip_runtime.h"

#include <cmath>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void single_embedding_kernel(float* state_space, float* x, int emb_dim, int a, unsigned int tau)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = tid / emb_dim;
	int j = tid % emb_dim;
	state_space[tid] = x[(a - j) * tau + i];
}

__global__ void joint_embedding_kernel(float* state_space, float* x, float* y, int emb_dim, int a, unsigned int tau)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = tid / emb_dim;
	int j = tid % emb_dim;
	int time = (a - j) * tau + i;
	state_space[tid] = x[time] + y[time];
}

hipError_t dev_alloc_timeseries(float** dev_x, float** dev_y, int dev_x_size) {
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_x, dev_x_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)dev_y, dev_x_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

hipError_t dev_copy_timeseries(float* dev_x, float* dev_y, float* host_x, float* host_y, int n) {
	// Copy input vector from host memory to GPU buffers.
	// Leave the pitch with memory garbage
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(dev_x, host_x, n * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed in dev_copy_timeseries!");
	}

	cudaStatus = hipMemcpy(dev_y, host_y, n * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed in dev_copy_timeseries!");
	}

	return cudaStatus;
}

hipError_t dev_alloc_manifolds(float** dev_state_space_X, float** dev_state_space_Y, float** dev_state_space_J, float** dev_state_space_Z, int dev_state_space_size) {
	// Allocate GPU buffers for the state space and time series
	// The kernel will over-index n, but that's not a problem, memory garbage can be used as pitch
	// This way branch divergence can not happen

	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_state_space_X, dev_state_space_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)dev_state_space_Y, dev_state_space_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)dev_state_space_J, dev_state_space_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)dev_state_space_Z, dev_state_space_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

hipError_t embed_manifolds(float** dev_state_space_X,
	                        float** dev_state_space_Y,
	                        float** dev_state_space_J,
	                        float** dev_state_space_Z,
                            int& dev_state_space_size,
	                        float* host_x,
                            float* host_y,
	                        int n,
                            int& dev_x_size,
	                        int emb_dim,
	                        int tau,
	                        int num_threads_per_block) {
	// time-series are row-like in memory, therefore the block layout in the grid, and the thread layout in a block should be row-like as well for better memory coalesce

	hipError_t cudaStatus;

	int a = emb_dim - 1;
	int offset = a * tau;
	int num_rows = n - offset;
	int num_elems = num_rows * emb_dim;

	int num_blocks = (num_elems + (num_threads_per_block - 1)) / num_threads_per_block;

	// Allocate GPU memory
	float* dev_x = 0;
	float* dev_y = 0;

	dev_state_space_size = num_blocks * num_threads_per_block;
	int i = dev_state_space_size / emb_dim;
	dev_x_size = offset + i;

	cudaStatus = dev_alloc_timeseries(&dev_x, &dev_y, dev_x_size);
	cudaStatus = dev_alloc_manifolds(dev_state_space_X, dev_state_space_Y, dev_state_space_J, dev_state_space_Z, dev_state_space_size);

	// Copy time-series to GPU memory
	cudaStatus = dev_copy_timeseries(dev_x, dev_y, host_x, host_y, n);

	// Launch the embedding kernels on the GPU
	single_embedding_kernel<<<num_blocks, num_threads_per_block>>>(*dev_state_space_X, dev_x, emb_dim, a, tau);
	single_embedding_kernel<<<num_blocks, num_threads_per_block>>>(*dev_state_space_Y, dev_y, emb_dim, a, tau);
	joint_embedding_kernel<<<num_blocks, num_threads_per_block>>>(*dev_state_space_J, dev_x, dev_y, emb_dim, a, tau);

	// Check for any errors launching the kernels
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "embedding kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching embedding_kernel!\n", cudaStatus);
	}

	hipFree(dev_x);
	hipFree(dev_y);
	return cudaStatus;
}



